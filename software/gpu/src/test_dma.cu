#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <GpuAsync.h>


//-----------------------------------------------------------------------------

void checkError(hipError_t status);
bool wasError(hipError_t status);

//-----------------------------------------------------------------------------

__global__ void data_move(uint32_t *in, uint32_t *out) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //int tid;

    uint32_t size = in[1]/4;

    if ( tid < size ) {
    //for (tid=0; tid < size; tid++) {
        out[tid] = in[tid+8];
    }
}


int main(int argc, char *argv[]) {
   uint x;
   int res = -1;

   int fd = open("/dev/datagpu_0", O_RDWR);
   if (fd < 0) {
      printf("Error open file\n");
      return -1;
   }

   ////////////////////////////////////////////
   // Open and setup GPU
   ////////////////////////////////////////////
   hipError_t status;
   checkError(hipInit(0));

   int total = 0;
   checkError(hipGetDeviceCount(&total));
   fprintf(stderr, "Total devices: %d\n", total);

   hipDevice_t device;
   checkError(hipDeviceGet(&device, 1));

   char name[256];
   checkError(hipDeviceGetName(name, 256, device));
   fprintf(stderr, "Select device: %s\n", name);

   size_t global_mem = 0;
   checkError( hipDeviceTotalMem(&global_mem, device));
   fprintf(stderr, "Global memory: %llu MB\n", (unsigned long long)(global_mem >> 20));
   if(global_mem > (unsigned long long)4*1024*1024*1024L) fprintf(stderr, "64-bit Memory Address support\n");

   hipCtx_t  context;
   checkError(hipCtxCreate(&context, 0, device));

   ////////////////////////////////////////////////
   // Create write and read buffers
   ////////////////////////////////////////////////
   size_t size = 0x10000;
   hipDeviceptr_t hwWritePtr = 0;
   hipDeviceptr_t hwReadPtr  = 0;
   uint32_t * hostWriteBuff = (uint32_t *)malloc(size);
   uint32_t * hostReadBuff  = (uint32_t *)malloc(size);

   memset(hostWriteBuff,0,size);
   memset(hostReadBuff,0,size);

   status = hipMalloc(&hwWritePtr, size);
   if(wasError(status)) printf("Failed to alloc write pointer\n");

   status = hipMalloc(&hwReadPtr, size);
   if(wasError(status)) printf("Failed to alloc read pointer\n");

   hipMemcpyHtoD( hwWritePtr, hostWriteBuff, size );
   hipMemcpyHtoD( hwReadPtr, hostReadBuff, size );

   uint32_t flag = 1;

   hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, hwWritePtr);
   hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, hwReadPtr);

   ////////////////////////////////////
   // Add buffer to hardware
   ////////////////////////////////////
   printf("Setting write pointer\n");
   gpuAddNvidiaMemory(fd,1,(uint64_t)hwWritePtr,size);
   printf("Setting read pointer\n");
   gpuAddNvidiaMemory(fd,0,(uint64_t)hwReadPtr,size);
   printf("Done with pointers\n");

   ////////////////////////////////////////////////
   // Map FPGA register space to GPU
   ////////////////////////////////////////////////

   // Setup FPGA Registers
   printf("Mapping FPGA registers\n");
   uint8_t * swFpgaRegs = (uint8_t *) dmaMapRegister(fd, 0x00A00000, 0x00100000);

   if ( swFpgaRegs == NULL ) printf("Failed to map FPGA registers\n");
   else printf("swFpgaRegs = 0x%lx\n",(uint64_t)swFpgaRegs);

   printf("Enabling IO memory for FPGA registers\n");
   status = hipHostRegister(swFpgaRegs, 0x00100000, hipHostRegisterIoMemory);
   if(wasError(status)) printf("Failed to host register memory. Status = %i\n",status);

   hipDeviceptr_t hwWriteStart = 0;
   hipDeviceptr_t hwReadStart  = 0;

   printf("Mapping write start register\n");
   status = hipHostGetDevicePointer(&hwWriteStart, swFpgaRegs + 0x300, 0);
   if(wasError(status)) printf("Failed to map device write start pointer. Status = %i\n",status);

   printf("Mapping read start register\n");
   status = hipHostGetDevicePointer(&hwReadStart,  swFpgaRegs + 0x400, 0);
   if(wasError(status)) printf("Failed to map device read start pointer. Status = %i\n",status);

   printf("Mapped FPGA registers\n");

   ////////////////////////////////////
   // Setup GPU streaming
   ////////////////////////////////////
   hipStream_t stream;

   hipStreamCreate(&stream);

   fprintf(stderr, "Trigger write\n");
   hipStreamWriteValue32(stream,hwWriteStart,0x00,0);
   hipStreamWaitValue32(stream, hwWritePtr+4, 1, hipStreamWaitValueGte);

   // Do GPU processing here
   data_move<<<4,1024,1,stream>>>((uint32_t*)hwWritePtr,(uint32_t*)hwReadPtr);
   //data_move<<<1,1,1,stream>>>((uint32_t*)hwWritePtr,(uint32_t*)hwReadPtr);

   //hipStreamWriteValue32(stream,hwReadStart,((uint32_t*)(hwWritePtr))[1],0);
   hipStreamWriteValue32(stream,hwReadStart,0x2020,0);

   fprintf(stderr, "Stream Sync\n");
   hipStreamSynchronize(stream);

   hipCtxSynchronize();
   hipMemcpyDtoH( hostWriteBuff, hwWritePtr, size );
   hipMemcpyDtoH( hostReadBuff, hwReadPtr, size );

   for (x=0; x < 100; x++) printf("data: %i 0x%8x - 0x%8x\n",x,hostWriteBuff[x],hostReadBuff[x]);

   res = gpuRemNvidiaMemory(fd);
   if(res < 0) fprintf(stderr, "Error in IOCTL_GPUDMA_MEM_UNLOCK\n");
}

// -------------------------------------------------------------------

void checkError(hipError_t status)
{
    if(status != hipSuccess) {
        const char *perrstr = 0;
        hipError_t ok = hipDrvGetErrorString(status,&perrstr);
        if(ok == hipSuccess) {
            if(perrstr) {
                fprintf(stderr, "info: %s\n", perrstr);
            } else {
                fprintf(stderr, "info: unknown error\n");
            }
        }
        exit(0);
    }
}

//-----------------------------------------------------------------------------

bool wasError(hipError_t status)
{
    if(status != hipSuccess) {
        const char *perrstr = 0;
        hipError_t ok = hipDrvGetErrorString(status,&perrstr);
        if(ok == hipSuccess) {
            if(perrstr) {
                fprintf(stderr, "info: %s\n", perrstr);
            } else {
                fprintf(stderr, "info: unknown error\n");
            }
        }
        return true;
    }
    return false;
}

